#include "hip/hip_runtime.h"
#include "mosc.h"
#include "mosc3.h"
#include <stdio.h>

#define elec (0)
#define muon (1)
#define tau  (2)
#define re (0)
#define im (1)

//#define ZERO_CP
static int matrixtype = standard_type;

/* Flag to tell us if we're doing nu_e or nu_sterile matter effects */
//static NuType matterFlavor = nue_type;


/***********************************************************************
  getM
  Compute the matter-mass vector M, dM = M_i-M_j and
  and dMimj. type<0 means anti-neutrinos type>0 means "real" neutrinos
***********************************************************************/


__device__ void getHVac(fType Enu, fType rho,
                        fType Mix[][3][2], fType dmVacVac[][3], int antitype,
                        fType HVac[][3][2])
{
  fType dmVacDiag[3][3][2], MixConjTranspose[3][3][2], tmp[3][3][2];
  clear_complex_matrix(HVac);
  clear_complex_matrix(dmVacDiag);
  dmVacDiag[1][1][re] = dmVacVac[1][0]/(2*Enu);
  dmVacDiag[2][2][re] = dmVacVac[2][0]/(2*Enu);
  clear_complex_matrix(tmp);
  conjugate_transpose_complex_matrix(Mix, MixConjTranspose);
  multiply_complex_matrix(dmVacDiag, MixConjTranspose, tmp);
  multiply_complex_matrix(Mix, tmp, HVac);
}

__device__ void getHNSI(fType rho, fType NSIEps[][3], int antitype, fType HNSI[][3][2])
{
  fType tworttwoGf = 1.52588e-4;
  fType fact = 3.0*rho*tworttwoGf/2.0; // assume 3x electron density for
                                     // "NSI"-quark (e.g., d) density
  if (antitype<0) fact =  -fact; /* Anti-neutrinos */
  else        fact = fact; /* Neutrinos */
  for (int i=0; i<3; i++) {
    for (int j=0; j<3; j++) {
        HNSI[i][j][0] = fact*NSIEps[i][j];
        HNSI[i][j][1] = 0.0; // only real NSI for now
    }
  }
}

__device__ void getHMatMassEigenstateBasis(fType Mix[][3][2], fType HMat[][3][2],
                                           fType HMatMassEigenstateBasis[][3][2])
{
    fType MixConjTranspose[3][3][2], tmp[3][3][2];
    clear_complex_matrix(MixConjTranspose);
    clear_complex_matrix(tmp);
    conjugate_transpose_complex_matrix(Mix, MixConjTranspose);
    multiply_complex_matrix(HMat, Mix, tmp);
    multiply_complex_matrix(MixConjTranspose, tmp, HMatMassEigenstateBasis);
}


__device__ void getHMat(fType Enu, fType rho,
                        fType Mix[][3][2], fType NSIEps[][3],
                        fType dmVacVac[][3], int antitype,
                        fType HMat[][3][2])
{
  fType HSI[3][3][2], HNSI[3][3][2];
  fType tworttwoGf = 1.52588e-4;
  fType a = rho*tworttwoGf/2., MatParam = 0.0;
  clear_complex_matrix(HSI); clear_complex_matrix(HNSI);
  if (antitype<0) MatParam =  -a; /* Anti-neutrinos */
  else        MatParam = a; /* Neutrinos */
  HSI[0][0][re] = MatParam;
  getHNSI(rho, NSIEps, antitype, HNSI);
  // This is where the non-standard matter interaction Hamiltonian is added to
  // the standard matter Hamiltonian
  add_complex_matrix(HSI, HNSI, HMat);
}


__device__ void getM(fType Enu, fType rho,
                     fType Mix[][3][2], fType dmVacVac[][3], int antitype,
                     fType dmMatMat[][3], fType dmMatVac[][3],
                     fType HMat[][3][2])
{
  //fType c0[2], c1[2], c2[2];
  int i,j,k;
  fType c0, c1, c2, c0_final, c1_final, c2_final;
  fType c0V, c1V, c2V;
  fType p, q, pV, qV;
  fType arg, theta0, theta1, theta2, tmp, M1Sq, M2Sq, M3Sq;
  fType argV, theta0V, theta1V, theta2V, tmpV, M1SqV, M2SqV, M3SqV;

  fType mMatU[3], mMatV[3], mMat[3];
  fType HEEHMuMuHTauTau;
  fType HMuTauModulusSq, HETauModulusSq, HEMuModulusSq, ReHEMuHMuTauHTauE;
  // following only here temporarily
  fType tworttwoGf = 1.52588e-4;
  fType a = rho*tworttwoGf/2.0, MatParam = 0.0;
  if (antitype<0) MatParam =  -a; // Anti-neutrinos
  else        MatParam = a; // Neutrinos
  

#ifndef ZERO_CP

  ReHEMuHMuTauHTauE = HMat[elec][muon][re]*(HMat[muon][tau][re]*HMat[tau][elec][re] -
                                            HMat[muon][tau][im]*HMat[tau][elec][im]) -
    HMat[elec][muon][im]*(HMat[muon][tau][im]*HMat[tau][elec][re] + HMat[muon][tau][re]*HMat[tau][elec][im]);

  HEMuModulusSq = HMat[elec][muon][re]*HMat[elec][muon][re] + HMat[elec][muon][im]*HMat[elec][muon][im];
  HETauModulusSq = HMat[elec][tau][re]*HMat[elec][tau][re] + HMat[elec][tau][im]*HMat[elec][tau][im];
  HMuTauModulusSq = HMat[muon][tau][re]*HMat[muon][tau][re] + HMat[muon][tau][im]*HMat[muon][tau][im];

  HEEHMuMuHTauTau = HMat[elec][elec][re]*(HMat[muon][muon][re]*HMat[tau][tau][re] -
                                            HMat[muon][muon][im]*HMat[tau][tau][im]) -
    HMat[elec][elec][im]*(HMat[muon][muon][im]*HMat[tau][tau][re] + HMat[muon][muon][re]*HMat[tau][tau][im]);


  //c1 = H_{ee}H_{\mu\mu} + H_{ee}H_{\tau\tau} + H_{\mu\mu}H_{\tau\tau} - |H_{e\mu}|^2
  //     - |H_{\mu\tau}|^2 - |H_{e\tau}|^2


  c1_final = HMat[elec][elec][re]*(HMat[muon][muon][re] + HMat[tau][tau][re]) -
           HMat[elec][elec][im]*(HMat[muon][muon][im] + HMat[tau][tau][im]) +
           HMat[muon][muon][re]*HMat[tau][tau][re] - HMat[muon][muon][im]*HMat[tau][tau][im] -
           HEMuModulusSq - HMuTauModulusSq - HETauModulusSq;

#else

  ReHEMuHMuTauHTauE = HMat[elec][muon][re]*(HMat[muon][tau][re]*HMat[tau][elec][re]);

  HEMuModulusSq = HMat[elec][muon][re]*HMat[elec][muon][re];
  HETauModulusSq = HMat[elec][tau][re]*HMat[elec][tau][re];
  HMuTauModulusSq = HMat[muon][tau][re]*HMat[muon][tau][re];

  HEEHMuMuHTauTau = HMat[elec][elec][re]*(HMat[muon][muon][re]*HMat[tau][tau][re]);

  c1_final = HMat[elec][elec][re]*(HMat[muon][muon][re] + HMat[tau][tau][re]) +
           HMat[muon][muon][re]*HMat[tau][tau][re] -
           HEMuModulusSq - HMuTauModulusSq - HETauModulusSq;

  //printf("Mix[im]: %.10f \n", Mix[0][2][im]);
  //printf("Enu: %.10f \n", Enu);
#endif
  //c0 = H_{ee}|H_{\mu\tau}|^2 + H_{\mu\mu}|H_{e\tau}|^2 + H_{\tau\tau}|H_{e\mu}|^2
  //     - 2Re(H_{e\mu}H_{\mu\tau}H_{\tau e}) - H_{ee}H_{\mu\mu}H_{\tau\tau}

  c0_final = HMat[elec][elec][re]*HMuTauModulusSq + HMat[muon][muon][re]*HETauModulusSq +
    HMat[tau][tau][re]*HEMuModulusSq - 2.0*ReHEMuHMuTauHTauE - HEEHMuMuHTauTau;

  //c2 = -H_{ee} - H_{\mu\mu} - H_{\tau\tau}

  c2_final = -HMat[elec][elec][re] - HMat[muon][muon][re] - HMat[tau][tau][re];

  //printf("rho, c0_num, c1_num, c2_num: %.5f %.10f %.10f %.10f \n", rho, c0_final, c1_final, c2_final);
  c0V = 0.0;
  c1V = (1.0/(2.0*Enu*2.0*Enu))*(dmVacVac[1][0]*dmVacVac[2][0]);
  c2 = (-1.0/(2.0*Enu))*(2.0*Enu*MatParam + dmVacVac[1][0] + dmVacVac[2][0]);
  c2V = (-1.0/(2.0*Enu))*(dmVacVac[1][0] + dmVacVac[2][0]);
#ifndef ZERO_CP
  c0 = (-1.0/(2.0*Enu*2.0*Enu*2.0*Enu))*2.0*Enu*MatParam*dmVacVac[1][0]*dmVacVac[2][0]*
                (Mix[0][0][re]*Mix[0][0][re] + Mix[0][0][im]*Mix[0][0][im]);

  c1 = (1.0/(2.0*Enu*2.0*Enu))*(dmVacVac[1][0]*dmVacVac[2][0] + 2.0*Enu*MatParam*
                (dmVacVac[1][0]*(1.0 - (Mix[0][1][re]*Mix[0][1][re] + 
                                      Mix[0][1][im]*Mix[0][1][im])
                                ) +
                dmVacVac[2][0]*(1.0 - (Mix[0][2][re]*Mix[0][2][re] + 
                                      Mix[0][2][im]*Mix[0][2][im])))
       );

#else
  c0 = (-1.0/(2.0*Enu*2.0*Enu*2.0*Enu))*2.0*Enu*MatParam*dmVacVac[1][0]*dmVacVac[2][0]*
                (Mix[0][0][re]*Mix[0][0][re]);
  c1 = (1.0/(2.0*Enu*2.0*Enu))*(dmVacVac[1][0]*dmVacVac[2][0] + 2.0*Enu*MatParam*
                (dmVacVac[1][0]*(1.0 - (Mix[0][1][re]*Mix[0][1][re])
                                ) +
                dmVacVac[2][0]*(1.0 - (Mix[0][2][re]*Mix[0][2][re])))
       );
#endif

                
  //printf("rho, c0, c1, c2: %.5f %.10f %.10f %.10f \n",rho,c0,c1,c2);

  p = c2_final*c2_final - 3.0*c1_final;
  pV = (1.0/(2.0*Enu*2.0*Enu))*(dmVacVac[1][0]*dmVacVac[1][0] +
                              dmVacVac[2][0]*dmVacVac[2][0] - 
                              dmVacVac[1][0]*dmVacVac[2][0]);
  if (p<0.0) {
      printf("getM: p < 0 ! \n");
      p = 0.0;
  }
  
  q = -27.0*c0_final/2.0 - c2_final*c2_final*c2_final + 9.0*c1_final*c2_final/2.0;
  qV = (1.0/(2.0*Enu*2.0*Enu*2.0*Enu))*(
        (dmVacVac[1][0] + dmVacVac[2][0])*(dmVacVac[1][0] + dmVacVac[2][0])*
        (dmVacVac[1][0] + dmVacVac[2][0]) - (9.0/2.0)*dmVacVac[1][0]*dmVacVac[2][0]*
        (dmVacVac[1][0] + dmVacVac[2][0]));

  tmp = p*p*p - q*q;
  tmpV = pV*pV*pV - qV*qV;
  if (tmp<0.0) {
    printf("getM: p^3 - q^2 < 0 !\n");
    tmp = 0.0;
  }
  theta0 = theta1 = theta2 = atan2(sqrt(tmp), q)/3.0;
  theta0V = theta1V = theta2V = atan2(sqrt(tmpV), qV)/3.0;
  theta0 += (2.0/3.0)*M_PI;
  theta0V += (2.0/3.0)*M_PI;
  theta1 -= (2.0/3.0)*M_PI;
  theta1V -= (2.0/3.0)*M_PI;
  //printf("theta0, theta1, theta2: %.10f %.10f %.10f \n", theta0, theta1, theta2);
  // add dmVacVac[0][0]?
  M1Sq = 2.0*Enu*((2.0/3.0)*sqrt(p)*cos(theta0) - c2_final/3.0 + dmVacVac[0][0]);
  M2Sq = 2.0*Enu*((2.0/3.0)*sqrt(p)*cos(theta1) - c2_final/3.0 + dmVacVac[0][0]);
  M3Sq = 2.0*Enu*((2.0/3.0)*sqrt(p)*cos(theta2) - c2_final/3.0 + dmVacVac[0][0]);
  M1SqV = 2.0*Enu*((2.0/3.0)*sqrt(pV)*cos(theta0V) - c2V/3.0 + dmVacVac[0][0]);
  M2SqV = 2.0*Enu*((2.0/3.0)*sqrt(pV)*cos(theta1V) - c2V/3.0 + dmVacVac[0][0]);
  M3SqV = 2.0*Enu*((2.0/3.0)*sqrt(pV)*cos(theta2V) - c2V/3.0 + dmVacVac[0][0]);

  mMatU[0] = M1Sq;
  mMatU[1] = M2Sq;
  mMatU[2] = M3Sq;
  mMatV[0] = M1SqV;
  mMatV[1] = M2SqV;
  mMatV[2] = M3SqV;
  //printf("m1sq, m2sq, m3sq: %.10f %.10f %.10f \n",M1Sq,M2Sq,M3Sq);
  //printf("m1sqV, m2sqV, m3sqV: %.10f %.10f %.10f \n",M1SqV,M2SqV,M3SqV);

  /* Sort according to which reproduce the vaccum eigenstates */
  for (i=0; i<3; i++) {
    tmpV = fabs(dmVacVac[i][0]-mMatV[0]);
    k = 0;
    for (j=1; j<3; j++) {
      tmp = fabs(dmVacVac[i][0]-mMatV[j]);
      if (tmp<tmpV) {
        k = j;
        tmpV = tmp;
      }
    }
    mMat[i] = mMatU[k];
  }
  for (i=0; i<3; i++) {
    for (j=0; j<3; j++) {
      dmMatMat[i][j] = mMat[i] - mMat[j];
      dmMatVac[i][j] = mMat[i] - dmVacVac[j][0];
    }
  }
 /*
 if (antitype < 0){
    printf("rho, m1, m2, m3, %.5f, %.10f, %.10f, %.10f, \n", rho, dmMatMat[0][0], dmMatMat[0][1], dmMatMat[0][2]);
    printf("rho, m1V, m2V, m3V, %.5f, %.10f, %.10f, %.10f, \n", rho, dmVacVac[0][0], dmVacVac[0][1], dmVacVac[0][2]);
 }
 */
}
  


__device__ void getMBarger(fType Enu, fType rho,
                     fType Mix[][3][2], fType dmVacVac[][3], int antitype,
                     fType dmMatMat[][3], fType dmMatVac[][3])
{
  int i, j, k;
  fType alpha, beta, gamma, fac=0.0, arg, tmp;
  fType alphaV, betaV, gammaV, argV, tmpV;
  fType theta0, theta1, theta2;
  fType theta0V, theta1V, theta2V;
  fType mMatU[3], mMatV[3], mMat[3];
  fType tworttwoGf = 1.52588e-4;

  /* Equations (22) fro Barger et.al.*/
  /* Reverse the sign of the potential depending on neutrino type */
  //if (matterFlavor == nue_type) {
  /* If we're doing matter effects for electron neutrinos */
  if (antitype<0) fac =  tworttwoGf*Enu*rho; /* Anti-neutrinos */
  else        fac = -tworttwoGf*Enu*rho; /* Real-neutrinos */
  //}
  //else if (matterFlavor == sterile_type) {
  /* If we're doing matter effects for sterile neutrinos */
  //if (antitype<0) fac = -0.5*tworttwoGf*Enu*rho; /* Anti-neutrinos */

  //   else        fac =  0.5*tworttwoGf*Enu*rho; /* Real-neutrinos */
  // }
  /* The strategy to sort out the three roots is to compute the vacuum
   * mass the same way as the "matter" masses are computed then to sort
   * the results according to the input vacuum masses
   */

  alpha  = fac + dmVacVac[0][1] + dmVacVac[0][2];
  alphaV = dmVacVac[0][1] + dmVacVac[0][2];

#ifndef ZERO_CP
  beta = dmVacVac[0][1]*dmVacVac[0][2] +
    fac*(dmVacVac[0][1]*(1.0 - Mix[elec][1][re]*Mix[elec][1][re] -
                         Mix[elec][1][im]*Mix[elec][1][im]) +
         dmVacVac[0][2]*(1.0 - Mix[elec][2][re]*Mix[elec][2][re] -
                         Mix[elec][2][im]*Mix[elec][2][im]));
  betaV = dmVacVac[0][1]*dmVacVac[0][2];

#else
  beta = dmVacVac[0][1]*dmVacVac[0][2] +
    fac*(dmVacVac[0][1]*(1.0 - Mix[elec][1][re]*Mix[elec][1][re]) +
         dmVacVac[0][2]*(1.0- Mix[elec][2][re]*Mix[elec][2][re]));
  betaV = dmVacVac[0][1]*dmVacVac[0][2];
#endif

#ifndef ZERO_CP
  gamma = fac*dmVacVac[0][1]*dmVacVac[0][2]*
    (Mix[elec][0][re]*Mix[elec][0][re]+Mix[elec][0][im]*Mix[elec][0][im]);
  gammaV = 0.0;
#else
  gamma = fac*dmVacVac[0][1]*dmVacVac[0][2]*
    (Mix[elec][0][re]*Mix[elec][0][re]);
  gammaV = 0.0;
#endif

  //printf("alpha, beta, gamma: %.10f %.10f %.10f \n", alpha, beta, gamma);

  /* Compute the argument of the arc-cosine */
  tmp = alpha*alpha-3.0*beta;
  tmpV = alphaV*alphaV-3.0*betaV;
  if (tmp<0.0) {
    printf("getM: alpha^2-3*beta < 0 !\n");
    tmp = 0.0;
  }

  /* Equation (21) */
  arg = (2.0*alpha*alpha*alpha-9.0*alpha*beta+27.0*gamma)/
    (2.0*sqrt(tmp*tmp*tmp));
  if (fabs(arg)>1.0) arg = arg/fabs(arg);
  argV = (2.0*alphaV*alphaV*alphaV-9.0*alphaV*betaV+27.0*gammaV)/
    (2.0*sqrt(tmpV*tmpV*tmpV));
  if (fabs(argV)>1.0) argV = argV/fabs(argV);

  /* These are the three roots the paper refers to */
  theta0 = acos(arg)/3.0;
  theta1 = theta0-(2.0*M_PI/3.0);
  theta2 = theta0+(2.0*M_PI/3.0);
  theta0V = acos(argV)/3.0;
  theta1V = theta0V-(2.0*M_PI/3.0);
  theta2V = theta0V+(2.0*M_PI/3.0);

  mMatU[0] = mMatU[1] = mMatU[2] = -(2.0/3.0)*sqrt(tmp);
  mMatU[0] *= cos(theta0); mMatU[1] *= cos(theta1); mMatU[2] *= cos(theta2);

  tmp = dmVacVac[0][0] - alpha/3.0;
  mMatU[0] += tmp; mMatU[1] += tmp; mMatU[2] += tmp;
  mMatV[0] = mMatV[1] = mMatV[2] = -(2.0/3.0)*sqrt(tmpV);
  mMatV[0] *= cos(theta0V); mMatV[1] *= cos(theta1V); mMatV[2] *= cos(theta2V);
  tmpV = dmVacVac[0][0] - alphaV/3.0;

  mMatV[0] += tmpV; mMatV[1] += tmpV; mMatV[2] += tmpV;

  /* Sort according to which reproduce the vaccum eigenstates */
  for (i=0; i<3; i++) {
    tmpV = fabs(dmVacVac[i][0]-mMatV[0]);
    k = 0;
    for (j=1; j<3; j++) {
      tmp = fabs(dmVacVac[i][0]-mMatV[j]);
      if (tmp<tmpV) {
        k = j;
        tmpV = tmp;
      }
    }
    mMat[i] = mMatU[k];
  }

  for (i=0; i<3; i++) {
    for (j=0; j<3; j++) {
      dmMatMat[i][j] = mMat[i] - mMat[j];
      dmMatVac[i][j] = mMat[i] - dmVacVac[j][0];
    }
 }
}

/***********************************************************************
 getANew (take into account generic potential matrix (=Hamiltonian))
 Calculate the transition amplitude matrix A (equation 10)
***********************************************************************/
__device__ void getANew(fType L, fType E, fType rho,
                        fType Mix[][3][2], fType dmMatVac[][3],
                        fType dmMatMat[][3], int antitype, fType HMatMassEigenstateBasis[][3][2],
                        fType A[3][3][2],
                        fType phase_offset)
{

  //int n, m, i, j, k;
  fType /*fac=0.0,*/ arg, c, s;
  // TCA ADDITION: set equal to 0!
  fType X[3][3][2] = {0.0};
  fType product[3][3][3][2] = {0.0};
  /* (1/2)*(1/(h_bar*c)) in units of GeV/(eV^2-km) */
  const fType LoEfac = 2.534;

  if ( phase_offset==0.0 )
    {
      get_productNew(L, E, rho, Mix, dmMatVac, dmMatMat, antitype, HMatMassEigenstateBasis,
                     product);
    }

  /////////////// product is JUNK /////////////

  /* Make the sum with the exponential factor */
  //hipMemset(X, 0, 3*3*2*sizeof(fType));
  //memset(X, 0, 3*3*2*sizeof(fType));
  for (int k=0; k<3; k++)
    {
      arg = -LoEfac*dmMatVac[k][0]*L/E;
      if ( k==2 ) arg += phase_offset ;
      c = cos(arg);
      s = sin(arg);
      for (int i=0; i<3; i++)
        {
          for (int j=0; j<3; j++)
            {
#ifndef ZERO_CP
              X[i][j][re] += c*product[i][j][k][re] - s*product[i][j][k][im];
              X[i][j][im] += c*product[i][j][k][im] + s*product[i][j][k][re];
#else
              X[i][j][re] += c*product[i][j][k][re];
              X[i][j][im] += s*product[i][j][k][re];
#endif
            }
        }
    }


  /* Compute the product with the mixing matrices */
  for(int i=0; i < 3; i++)
    for(int j = 0; j < 3; j++)
      for(int k = 0; k < 2; k++)
        A[i][j][k] = 0;

  for (int n=0; n<3; n++) {
    for (int m=0; m<3; m++) {
      for (int i=0; i<3; i++) {
        for (int j=0; j<3; j++) {
#ifndef ZERO_CP
          A[n][m][re] +=
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][re] +
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][im] +
            Mix[n][i][im]*X[i][j][re]*Mix[m][j][im] -
            Mix[n][i][im]*X[i][j][im]*Mix[m][j][re];
          //printf("\nregret %f %f %f",Mix[n][i][re], X[i][j][im], Mix[m][j][im]);
          A[n][m][im] +=
            Mix[n][i][im]*X[i][j][im]*Mix[m][j][im] +
            Mix[n][i][im]*X[i][j][re]*Mix[m][j][re] +
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][re] -
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][im];
#else
          A[n][m][re] +=
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][re];
          A[n][m][im] +=
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][re];
#endif
          //printf("\n %i %i %i A %f", n, m, re, A[n][m][re]);
        }
      }
    }
  }

  //printf("(getA) Aout: %f\n",A[0][0][0]);

}


__device__ void get_productNew(fType L, fType E, fType rho,fType Mix[][3][2],
                               fType dmMatVac[][3], fType dmMatMat[][3],
                               int antitype, fType HMatMassEigenstateBasis[][3][2],
                               fType product[][3][3][2])
{

  fType fac=0.0;
  fType twoEHmM[3][3][3][2];
  fType tworttwoGf = 1.52588e-4;

  /* (1/2)*(1/(h_bar*c)) in units of GeV/(eV^2-km) */
  /* Reverse the sign of the potential depending on neutrino type */
  //if (matterFlavor == nue_type) {

  /* If we're doing matter effects for electron neutrinos */
  if (antitype<0) fac =  tworttwoGf*E*rho; /* Anti-neutrinos */
  else        fac = -tworttwoGf*E*rho; /* Real-neutrinos */
  //  }

  /*
      else if (matterFlavor == sterile_type) {
      // If we're doing matter effects for sterile neutrinos
      if (antitype<0) fac = -0.5*tworttwoGf*E*rho; // Anti-neutrinos
      else        fac =  0.5*tworttwoGf*E*rho; // Real-neutrinos
      } */

  /* Calculate the matrix 2EH-M_j */
  for (int n=0; n<3; n++) {
    for (int m=0; m<3; m++) {

      twoEHmM[n][m][0][re] = 2.0*E*HMatMassEigenstateBasis[n][m][re];
#ifndef ZERO_CP

      twoEHmM[n][m][0][im] = 2.0*E*HMatMassEigenstateBasis[n][m][im];

#else

      twoEHmM[n][m][0][im] = 0.0 ;

#endif

      twoEHmM[n][m][1][re] = twoEHmM[n][m][2][re] = twoEHmM[n][m][0][re];
      twoEHmM[n][m][1][im] = twoEHmM[n][m][2][im] = twoEHmM[n][m][0][im];

      if (n==m) for (int j=0; j<3; j++)
                  twoEHmM[n][m][j][re] -= dmMatVac[j][n];
    }
  }

  /* Calculate the product in eq.(10) of twoEHmM for j!=k */
  for (int i=0; i<3; i++) {
    for (int j=0; j<3; j++) {
      for (int k=0; k<3; k++) {

#ifndef ZERO_CP

        product[i][j][0][re] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][re] -
          twoEHmM[i][k][1][im]*twoEHmM[k][j][2][im];
        product[i][j][0][im] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][im] +
          twoEHmM[i][k][1][im]*twoEHmM[k][j][2][re];
        product[i][j][1][re] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][re] -
          twoEHmM[i][k][2][im]*twoEHmM[k][j][0][im];
        product[i][j][1][im] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][im] +
          twoEHmM[i][k][2][im]*twoEHmM[k][j][0][re];
        product[i][j][2][re] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][re] -
          twoEHmM[i][k][0][im]*twoEHmM[k][j][1][im];
        product[i][j][2][im] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][im] +
          twoEHmM[i][k][0][im]*twoEHmM[k][j][1][re];

#else
        product[i][j][0][re] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][re];
        product[i][j][1][re] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][re];
        product[i][j][2][re] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][re];

#endif
      }
#ifndef ZERO_CP

      product[i][j][0][re] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][0][im] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][1][re] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][1][im] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][2][re] /= (dmMatMat[2][0]*dmMatMat[2][1]);
      product[i][j][2][im] /= (dmMatMat[2][0]*dmMatMat[2][1]);

#else
      product[i][j][0][re] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][1][re] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][2][re] /= (dmMatMat[2][0]*dmMatMat[2][1]);

#endif
    }
  }
}

/***********************************************************************
 getA
 Calculate the transition amplitude matrix A (equation 10)
***********************************************************************/
__device__ void getA(fType L, fType E, fType rho,
                     fType Mix[][3][2], fType dmMatVac[][3],
                     fType dmMatMat[][3], int antitype, fType A[3][3][2],
                     fType phase_offset)
{

  /*
    DARN - looks like this is all junk...more debugging needed...
  */

  //int n, m, i, j, k;
  fType /*fac=0.0,*/ arg, c, s;
  // TCA ADDITION: set equal to 0!
  fType X[3][3][2] = {0.0};
  fType product[3][3][3][2] = {0.0};
  /* (1/2)*(1/(h_bar*c)) in units of GeV/(eV^2-km) */
  const fType LoEfac = 2.534;

  if ( phase_offset==0.0 )
    {
      get_product(L, E, rho, Mix, dmMatVac, dmMatMat, antitype, product);
    }

  /////////////// product is JUNK /////////////

  for (int i=0; i<3; i++){
    for (int j=0; j<3; j++) {
  //printf(" product[%d][%d]: %f, %f\n",i,j,*product[i][j][0],*product[i][j][1]);
  //printf(" A[%d][%d]: %f, %f\n",i,j,A[i][j][0],A[i][j][1]);
    }
  }

  /* Make the sum with the exponential factor */
  //hipMemset(X, 0, 3*3*2*sizeof(fType));
  //memset(X, 0, 3*3*2*sizeof(fType));
  for (int k=0; k<3; k++)
    {
      arg = -LoEfac*dmMatVac[k][0]*L/E;
      if ( k==2 ) arg += phase_offset ;
      c = cos(arg);
      s = sin(arg);
      for (int i=0; i<3; i++)
        {
          for (int j=0; j<3; j++)
            {
#ifndef ZERO_CP
              X[i][j][re] += c*product[i][j][k][re] - s*product[i][j][k][im];
              X[i][j][im] += c*product[i][j][k][im] + s*product[i][j][k][re];
#else
              X[i][j][re] += c*product[i][j][k][re];
              X[i][j][im] += s*product[i][j][k][re];
#endif
            }
        }
    }


  /* Compute the product with the mixing matrices */
  for(int i=0; i < 3; i++)
    for(int j = 0; j < 3; j++)
      for(int k = 0; k < 2; k++)
        A[i][j][k] = 0;

  for (int n=0; n<3; n++) {
    for (int m=0; m<3; m++) {
      for (int i=0; i<3; i++) {
        for (int j=0; j<3; j++) {
#ifndef ZERO_CP
          A[n][m][re] +=
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][re] +
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][im] +
            Mix[n][i][im]*X[i][j][re]*Mix[m][j][im] -
            Mix[n][i][im]*X[i][j][im]*Mix[m][j][re];
          //printf("\nregret %f %f %f",Mix[n][i][re], X[i][j][im], Mix[m][j][im]);
          A[n][m][im] +=
            Mix[n][i][im]*X[i][j][im]*Mix[m][j][im] +
            Mix[n][i][im]*X[i][j][re]*Mix[m][j][re] +
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][re] -
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][im];
#else
          A[n][m][re] +=
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][re];
          A[n][m][im] +=
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][re];
#endif
          //printf("\n %i %i %i A %f", n, m, re, A[n][m][re]);
        }
      }
    }
  }

  //printf("(getA) Aout: %f\n",A[0][0][0]);

}


__device__ void get_product(fType L, fType E, fType rho,fType Mix[][3][2],
                            fType dmMatVac[][3], fType dmMatMat[][3],
                            int antitype,
                            fType product[][3][3][2])
{

  fType fac=0.0;
  fType twoEHmM[3][3][3][2];
  fType tworttwoGf = 1.52588e-4;

  /* (1/2)*(1/(h_bar*c)) in units of GeV/(eV^2-km) */
  /* Reverse the sign of the potential depending on neutrino type */
  //if (matterFlavor == nue_type) {

  /* If we're doing matter effects for electron neutrinos */
  if (antitype<0) fac =  tworttwoGf*E*rho; /* Anti-neutrinos */
  else        fac = -tworttwoGf*E*rho; /* Real-neutrinos */
  //  }

  /*
      else if (matterFlavor == sterile_type) {
      // If we're doing matter effects for sterile neutrinos
      if (antitype<0) fac = -0.5*tworttwoGf*E*rho; // Anti-neutrinos
      else        fac =  0.5*tworttwoGf*E*rho; // Real-neutrinos
      } */

  /* Calculate the matrix 2EH-M_j */
  for (int n=0; n<3; n++) {
    for (int m=0; m<3; m++) {

#ifndef ZERO_CP
      twoEHmM[n][m][0][re] =
        -fac*(Mix[0][n][re]*Mix[0][m][re]+Mix[0][n][im]*Mix[0][m][im]);
      twoEHmM[n][m][0][im] =
        -fac*(Mix[0][n][re]*Mix[0][m][im]-Mix[0][n][im]*Mix[0][m][re]);

      twoEHmM[n][m][1][re] = twoEHmM[n][m][2][re] = twoEHmM[n][m][0][re];
      twoEHmM[n][m][1][im] = twoEHmM[n][m][2][im] = twoEHmM[n][m][0][im];

#else

      twoEHmM[n][m][0][re] =
        -fac*(Mix[0][n][re]*Mix[0][m][re]);
      twoEHmM[n][m][0][im] = 0 ;
      twoEHmM[n][m][1][re] = twoEHmM[n][m][2][re] = twoEHmM[n][m][0][re];
      twoEHmM[n][m][1][im] = twoEHmM[n][m][2][im] = twoEHmM[n][m][0][im];

#endif

      if (n==m) for (int j=0; j<3; j++)
                  twoEHmM[n][m][j][re] -= dmMatVac[j][n];
    }
  }

  /* Calculate the product in eq.(10) of twoEHmM for j!=k */
  for (int i=0; i<3; i++) {
    for (int j=0; j<3; j++) {
      for (int k=0; k<3; k++) {

#ifndef ZERO_CP

        product[i][j][0][re] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][re] -
          twoEHmM[i][k][1][im]*twoEHmM[k][j][2][im];
        product[i][j][0][im] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][im] +
          twoEHmM[i][k][1][im]*twoEHmM[k][j][2][re];
        product[i][j][1][re] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][re] -
          twoEHmM[i][k][2][im]*twoEHmM[k][j][0][im];
        product[i][j][1][im] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][im] +
          twoEHmM[i][k][2][im]*twoEHmM[k][j][0][re];
        product[i][j][2][re] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][re] -
          twoEHmM[i][k][0][im]*twoEHmM[k][j][1][im];
        product[i][j][2][im] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][im] +
          twoEHmM[i][k][0][im]*twoEHmM[k][j][1][re];

#else
        product[i][j][0][re] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][re];
        product[i][j][1][re] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][re];
        product[i][j][2][re] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][re];

#endif
      }
#ifndef ZERO_CP

      product[i][j][0][re] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][0][im] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][1][re] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][1][im] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][2][re] /= (dmMatMat[2][0]*dmMatMat[2][1]);
      product[i][j][2][im] /= (dmMatMat[2][0]*dmMatMat[2][1]);

#else
      product[i][j][0][re] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][1][re] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][2][re] /= (dmMatMat[2][0]*dmMatMat[2][1]);

#endif
    }
  }
}
